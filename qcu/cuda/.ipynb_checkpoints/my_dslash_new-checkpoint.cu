#include "hip/hip_runtime.h"
#include "qcu.h"
#include <cstdio>
#include <time.h>
#include <cmath>
#define NC 3
#define ND 4
#define NS 4
#define BLOCK_SIZE 128

#define checkCudaErrors(err) { \
    if (err != hipSuccess) { \
        fprintf(stderr, "checkCudaErrors() API error = %04d \"%s\" from file <%s>, line %i.\n", \
                err, hipGetErrorString(err), __FILE__, __LINE__); \
                exit(-1); \
        }\
    }

#define getVecAddr(origin, x, y, z, t, Lx, Ly, Lz, Lt)  \
    ((origin) + (((t * Lz + z) *Ly + y)*Lx + x) * NS * NC)   // 9times
#define getGaugeAddr(origin, direction, x, y, z, t, Lx, Ly, Lz, Lt) \
    ((origin) + ((((direction * Lt + t)*Lz + z ) * Ly + y)*Lx + x) * NC * NC)    // 10 times

class Complex {
private:
    double real_;
    double imag_;
public:
    __device__ __host__
    Complex(double real, double imag) : real_(real), imag_(imag) { }
    __device__ __host__
    Complex() : real_(0), imag_(0) {}
    __device__ __host__
    Complex(const Complex& complex) : real_(complex.real_), imag_(complex.imag_){}
    __device__ __host__
    double norm2() {
        return sqrt(real_ * real_ + imag_ * imag_);
    }
    __device__ __host__
    void setImag(double imag) { imag_ = imag; }
    __device__ __host__
    void setReal(double real) { real_ = real; }
    __device__ __host__
    double real() const { return real_; }
    __device__ __host__
    double imag() const { return imag_; }

    __device__ __host__
    Complex& operator= (const Complex& complex) {
        real_ = complex.real_;
        imag_ = complex.imag_;
        return *this;
    }
    __device__ __host__
    Complex& operator= (double rhs) {
        real_ = rhs;
        imag_ = 0;
        return *this;
    }
    __device__ __host__
    Complex operator+(const Complex& complex) const {
        return Complex(real_+complex.real_, imag_+complex.imag_);
    }
    __device__ __host__
    Complex operator-(const Complex& complex) const {
        return Complex(real_-complex.real_, imag_-complex.imag_);
    }
    __device__ __host__
    Complex operator-() const{
        return Complex(-real_, -imag_);
    }
    __device__ __host__
    Complex operator*(const Complex& rhs) const {
        return Complex(real_ * rhs.real_ - imag_ * rhs.imag_, real_ * rhs.imag_ + imag_ * rhs.real_);
    }
    __device__ __host__
    Complex& operator*=(const Complex& rhs) {
        real_ = real_ * rhs.real_ - imag_ * rhs.imag_;
        imag_ = real_ * rhs.imag_ + imag_ * rhs.real_;
        return *this;
    }
    __device__ __host__
    Complex operator/ (const double& rhs) {
        return Complex(real_/rhs, imag_/rhs);
    }

    __device__ __host__
    Complex& operator+=(const Complex& rhs) {
        real_ += rhs.real_;
        imag_ += rhs.imag_;
        return *this;
    }

    __device__ __host__
    Complex& operator-=(const Complex& rhs) {
        real_ -= rhs.real_;
        imag_ -= rhs.imag_;
        return *this;
    }

    __device__ __host__
    Complex& clear2Zero() {
        real_ = 0;
        imag_ = 0;
        return *this;
    }
    __device__ __host__
    Complex conj() {
        return Complex(real_, -imag_);
    }
    __device__ __host__
    bool operator==(const Complex& rhs) {
        return real_ == rhs.real_ && imag_ == rhs.imag_;
    }
    __device__ __host__
    bool operator!=(const Complex& rhs) {
        return real_ != rhs.real_ || imag_ != rhs.imag_;
    }
    
};

__global__
void gpuDslash(void* U_ptr, void* a_ptr, void* b_ptr, int Lx, int Ly, int Lz, int Lt) {
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    // int tempn = thread;
    // // int t = thread / (Lx * Ly * Lz);
    // // int z = (thread - t * Lx * Ly * Lz) / (Lx * Ly);
    // // int y = (thread % (Lx * Ly)) / Lx;
    // // int x = thread % Lx;
    int t = thread / (Lx * Ly * Lz);
    thread -= t * (Lx * Ly * Lz);
    int z = thread / (Lx * Ly);
    thread -= z * (Lx * Ly);
    int y = thread / Lx;
    int x = thread - y * Lx;

    // int x = thread % Lx;
    // thread /= Lx;
    // int y = thread % Ly;
    // thread /= Ly;
    // int z = thread % Lz;
    // int t = thread / Lz;

    Complex *u;
    Complex *res;
    Complex *dest;
    Complex u_temp[NC * NC];            // for GPU
    Complex res_temp[NS * NC];          // for GPU
    Complex dest_temp[NS * NC];         // for GPU
    Complex u_last_line[NC];
    double norm;
    // Complex third_line_temp[NS];
    Complex temp;
    // #pragma unroll
    for (int i = 0; i < NS*NC; i++) {
        dest_temp[i].clear2Zero();
    }
    // memset(dest_temp, 0, sizeof(dest_temp));
    // \mu = 1
    dest = getVecAddr(static_cast<Complex*>(b_ptr), x, y, z, t, Lx, Ly, Lz, Lt);
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 0, x, y, z, t, Lx, Ly, Lz, Lt);
    // memcpy(u_temp, u, sizeof(u_temp));
    // #pragma unroll
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;

    res = getVecAddr(static_cast<Complex*>(a_ptr), (x+1)%Lx, y, z, t, Lx, Ly, Lz, Lt);
    // memcpy(res_temp, res, sizeof(res_temp));
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[3*NC+j] * Complex(0, 1)) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0,1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[2*NC+j] * Complex(0, 1)) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0,1);
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 0, (x+Lx-1)%Lx, y, z, t, Lx, Ly, Lz, Lt);
    // memcpy(u_temp, u, sizeof(u_temp));
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), (x+Lx-1)%Lx, y, z, t, Lx, Ly, Lz, Lt);
    // memcpy(res_temp, res, sizeof(res_temp));
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[3*NC+j] * Complex(0,1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0, -1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[2*NC+j] * Complex(0,1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0, -1);
        }
    }
    // \mu = 2
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 1, x, y, z, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, (y+1)%Ly, z, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[3*NC+j]) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[2*NC+j]) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += -temp;
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 1, x, (y+Ly-1)%Ly, z, t, Lx, Ly, Lz, Lt);
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, (y+Ly-1)%Ly, z, t, Lx, Ly, Lz, Lt);
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        // #pragma unroll
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[3*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[3*3+i] += -temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[2*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[2*3+i] += temp;
        }
    }
    // \mu = 3
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 2, x, y, z, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, (z+1)%Lz, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[2*NC+j] * Complex(0, 1)) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0, 1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[3*NC+j] * Complex(0,1)) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0, -1);
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 2, x, y, (z+Lz-1)%Lz, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, (z+Lz-1)%Lz, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[2*NC+j] * Complex(0, 1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += temp * Complex(0, -1);
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[3*NC+j] * Complex(0, 1)) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += temp * Complex(0, 1);
        }
    }
    // \mu = 4
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 3, x, y, z, t, Lx, Ly, Lz, Lt);
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, z, (t+1)%Lt, Lx, Ly, Lz, Lt);
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] - res_temp[2*NC+j]) * u_temp[i*NC+j];
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += -temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] - res_temp[3*NC+j]) * u_temp[i*NC+j];
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += -temp;
        }
    }
    u = getGaugeAddr(static_cast<Complex*>(U_ptr), 3, x, y, z, (t+Lt-1)%Lt, Lx, Ly, Lz, Lt);
    // #pragma unroll
    // for (int i = 0; i < NC * NC; i++) {
    //     u_temp[i] = u[i];
    // }
    for (int i = 0; i < 2 * NC; i++) {
        u_temp[i] = u[i];
    }
    u_last_line[0] = u_temp[1] * u_temp[5] - u_temp[2] * u_temp[4];
    u_last_line[1] = u_temp[2] * u_temp[3] - u_temp[0] * u_temp[5];
    u_last_line[2] = u_temp[0] * u_temp[4] - u_temp[1] * u_temp[3];
    norm = sqrt(u_last_line[0].norm2() * u_last_line[0].norm2() + u_last_line[1].norm2() * u_last_line[1].norm2() + u_last_line[2].norm2() * u_last_line[2].norm2());
    u_temp[6] = u_last_line[0].conj() / norm;
    u_temp[7] = u_last_line[1].conj() / norm;
    u_temp[8] = u_last_line[2].conj() / norm;
    res = getVecAddr(static_cast<Complex*>(a_ptr), x, y, z, (t+Lt-1)%Lt, Lx, Ly, Lz, Lt);
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        res_temp[i] = res[i];
    }
    // #pragma unroll
    for (int i = 0; i < NC; i++) {
        for (int j = 0; j < NC; j++) {
            // first row vector with col vector
            temp = (res_temp[0*NC+j] + res_temp[2*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[0*3+i] += temp;
            dest_temp[2*3+i] += temp;
            // second row vector with col vector
            temp = (res_temp[1*NC+j] + res_temp[3*NC+j]) * u_temp[j*NC+i].conj();   // transpose and conj
            dest_temp[1*3+i] += temp;
            dest_temp[3*3+i] += temp;
        }
    }
    // end, copy result to dest
    // #pragma unroll
    for (int i = 0; i < NS * NC; i++) {
        dest[i] = dest_temp[i];
    }
    // memcpy(dest, dest_temp, sizeof(dest_temp));
}


void dslashQcu(void *fermion_out, void *fermion_in, void *gauge, QcuParam *param) {
    clock_t start, end;
    clock_t big_start, big_end;
    int Lx = param->lattice_size[0];
    int Ly = param->lattice_size[1];
    int Lz = param->lattice_size[2];
    int Lt = param->lattice_size[3];

    void* d_u;
    void* d_a;
    void* d_b;
    unsigned long u_size = ND * Lt * Lz * Ly * Lx * NC * NC * sizeof(Complex);
    unsigned long vec_size = Lt * Lz * Ly * Lx * NS * NC * sizeof(Complex);
    int space = Lx * Ly * Lz * Lt;

    big_start = clock();
    checkCudaErrors(hipMalloc(&d_u, u_size));
    checkCudaErrors(hipMalloc(&d_a, vec_size));
    checkCudaErrors(hipMalloc(&d_b, vec_size));

    checkCudaErrors(hipMemcpy(d_u, gauge, u_size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_a, fermion_in, vec_size, hipMemcpyHostToDevice));

    dim3 gridDim(space/BLOCK_SIZE);
    dim3 blockDim(BLOCK_SIZE);

    start = clock();
    // 改为GPU版本 -- kernel function
    gpuDslash<<<gridDim, blockDim>>>(d_u, d_a, d_b, Lx, Ly, Lz, Lt);
    hipError_t err = hipGetLastError();
    checkCudaErrors(err);
    // 数据同步
    checkCudaErrors(hipDeviceSynchronize());
    end = clock();
    // 执行结果不改变U和a，所以这里不去进行u和a的cpy
    checkCudaErrors(hipMemcpy(fermion_out, d_b, vec_size, hipMemcpyDeviceToHost));
    // free memory
    checkCudaErrors(hipFree(d_u));
    checkCudaErrors(hipFree(d_a));
    checkCudaErrors(hipFree(d_b));
    big_end = clock();
    printf("total time: (with malloc free memcpy) : %lf\n", (double)(big_end - big_start) / CLOCKS_PER_SEC);
    printf("total time: (without malloc free memcpy) : %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
}
